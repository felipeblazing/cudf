/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdlib>
#include <iostream>
#include <fstream>
#include <vector>
#include <sys/stat.h>

#include "gtest/gtest.h"
#include "gmock/gmock.h"

#include <cudf.h>
#include <NVStrings.h>

bool checkFile(const char *fname)
{
	struct stat st;
	return (stat(fname, &st) ? 0 : 1);
}



// DESCRIPTION: Simple test internal helper class to transfer cudf column data
// from device to host for test comparisons and debugging/development
template <typename T>
class gdf_host_column
{
public:
	gdf_host_column() = delete;
	explicit gdf_host_column(gdf_column* const col)
	{
		m_hostdata = std::vector<T>(col->size);
		hipMemcpy(m_hostdata.data(), col->data, sizeof(T) * col->size, hipMemcpyDeviceToHost);
	}

	auto hostdata() const -> const auto&
	{
		return m_hostdata;
	}
	void print() const
	{
		for (size_t i = 0; i < m_hostdata.size(); ++i)
		{
			std::cout << "[" << i << "]: value=" << m_hostdata[i] << "\n";
		}
	}

private:
	std::vector<T> m_hostdata;
};


void compare_floats(gdf_host_column<double> host_values, std::vector<double> compare_values){
	for(size_t index = 0; index < compare_values.size(); index++){
		EXPECT_DOUBLE_EQ(host_values.hostdata()[index], compare_values[index] );
	}
}

TEST(gdf_csv_test, Simple)
{
	const char* fname	= "/tmp/CsvSimpleTest.csv";
	const char* names[]	= { "A", "B", "C", "D", "E", "F", "G", "H", "I", "J" };
	const char* types[]	= { "int32", "int32", "int32", "int32", "int32",
							"int32", "int32", "int32", "int32", "int32", };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile <<	"10,20,30,40,50,60,70,80,90,100\n"\
				"11,21,31,41,51,61,71,81,91,101\n"\
				"12,22,32,42,52,62,72,82,92,102\n"\
				"13,23,33,43,53,63,73,83,93,103\n";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols		= std::extent<decltype(names)>::value;
		args.names			= names;
		args.dtype			= types;
		args.delimiter		= ',';
		args.lineterminator = '\n';
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		auto firstCol = gdf_host_column<int32_t>(args.data[0]);
		auto sixthCol = gdf_host_column<int32_t>(args.data[5]);
		EXPECT_THAT(firstCol.hostdata(), ::testing::ElementsAre(10, 11, 12, 13));
		EXPECT_THAT(sixthCol.hostdata(), ::testing::ElementsAre(60, 61, 62, 63));
	}
}

TEST(gdf_csv_float_test, SimpleFloat)
{
	const char* fname	= "/tmp/CsvSimpleTest.csv";
	const char* names[]	= { "A", "B", "C", "D", "E", "F", "G", "H", "I", "J" };
	const char* types[]	= { "float64", "float64", "float64", "float64", "float64",
							"float64", "float64", "float64", "float64", "float64", };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile <<	"10.5,20,30,40,50,60.1,70,80,90,100\n"\
				"11,21,31,41,51,61,71,81,91,101\n"\
				"12,22,32,42,52,62,72,82,92,102\n"\
				"13.54,23,33,43,53,63,73,83,93,103\n";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols		= std::extent<decltype(names)>::value;
		args.names			= names;
		args.dtype			= types;
		args.delimiter		= ',';
		args.lineterminator = '\n';
		args.decimal='.';
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		auto firstCol = gdf_host_column<double>(args.data[0]);
		auto sixthCol = gdf_host_column<double>(args.data[5]);
		std::vector<double> test_values_first({10.5, 11, 12, 13.54});
		compare_floats(firstCol,test_values_first);
		std::vector<double> test_values_sixth({60.1, 61, 62, 63});
		compare_floats(sixthCol,test_values_sixth);

	}
}

TEST(gdf_csv_test, MortPerf)
{
	gdf_error error = GDF_SUCCESS;

	csv_read_arg	args;
	const int num_cols = 31;

    args.num_cols = num_cols;

    const char ** dnames = new const char *[num_cols] {
        "loan_id",
        "monthly_reporting_period",
        "servicer",
        "interest_rate",
        "current_actual_upb",
        "loan_age",
        "remaining_months_to_legal_maturity",
        "adj_remaining_months_to_maturity",
        "maturity_date",
        "msa",
        "current_loan_delinquency_status",
        "mod_flag",
        "zero_balance_code",
        "zero_balance_effective_date",
        "last_paid_installment_date",
        "foreclosed_after",
        "disposition_date",
        "foreclosure_costs",
        "prop_preservation_and_repair_costs",
        "asset_recovery_costs",
        "misc_holding_expenses",
        "holding_taxes",
        "net_sale_proceeds",
        "credit_enhancement_proceeds",
        "repurchase_make_whole_proceeds",
        "other_foreclosure_proceeds",
        "non_interest_bearing_upb",
        "principal_forgiveness_upb",
        "repurchase_make_whole_proceeds_flag",
        "foreclosure_principal_write_off_amount",
        "servicing_activity_indicator"
    };
    args.names = dnames;

    const char ** dtype = new const char *[num_cols] {
    		"int64",
    		"date",
    		"category",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"date",
    		"float64",
    		"category",
    		"category",
    		"category",
    		"date",
    		"date",
    		"date",
    		"date",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"category",
    		"float64",
    		"category"
        };

        args.dtype = dtype;

		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = (char *)("/tmp/Performance_2000Q1.txt");

	if (  checkFile(args.filepath_or_buffer))
	{
		args.delimiter 		= '|';
		args.lineterminator = '\n';
		args.delim_whitespace = 0;
		args.skipinitialspace = 0;
		args.skiprows 		= 0;
		args.skipfooter 	= 0;
		args.dayfirst 		= 0;
        args.mangle_dupe_cols=true;
        args.num_cols_out=0;

        args.use_cols_int       = NULL;
        args.use_cols_char      = NULL;
        args.use_cols_char_len  = 0;
        args.use_cols_int_len   = 0;


        args.names = NULL;
        args.dtype = NULL;


		error = read_csv(&args);
	}

	EXPECT_TRUE( error == GDF_SUCCESS );
}

TEST(gdf_csv_test, Strings)
{
	const char* fname	= "/tmp/CsvStringsTest.csv";
	const char* names[]	= { "line", "verse" };
	const char* types[]	= { "int32", "str" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << names[0] << ',' << names[1] << ',' << '\n';
	outfile << "10,abc def ghi" << '\n';
	outfile << "20,\"jkl mno pqr\"" << '\n';
	outfile << "30,stu \"\"vwx\"\" yz" << '\n';
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};	
		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols		= std::extent<decltype(names)>::value;
		args.names			= names;
		args.dtype			= types;
		args.delimiter		= ',';
		args.lineterminator = '\n';
		args.skiprows		= 1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		// No filtering of any columns
		EXPECT_EQ( args.num_cols_out, args.num_cols );

		// Check the parsed string column metadata
		ASSERT_EQ( args.data[1]->dtype, GDF_STRING );
		auto stringList = reinterpret_cast<NVStrings*>(args.data[1]->data);

		ASSERT_NE( stringList, nullptr );
		auto stringCount = stringList->size();
		ASSERT_EQ( stringCount, 3u );
		auto stringLengths = std::unique_ptr<int[]>{ new int[stringCount] };
		ASSERT_NE( stringList->len(stringLengths.get(), false), 0u );

		// Check the actual strings themselves
		auto strings = std::unique_ptr<char*[]>{ new char*[stringCount] };
		for (size_t i = 0; i < stringCount; ++i) {
			ASSERT_GT( stringLengths[i], 0 );
			strings[i] = new char[stringLengths[i]];
		}
		EXPECT_EQ( stringList->to_host(strings.get(), 0, stringCount), 0 );
		EXPECT_STREQ( strings[0], "abc def ghi" );
		EXPECT_STREQ( strings[1], "\"jkl mno pqr\"" );
		EXPECT_STREQ( strings[2], "stu \"\"vwx\"\" yz" );
		for (size_t i = 0; i < stringCount; ++i) {
			delete[] strings[i];
		}
	}
}

TEST(gdf_csv_test, QuotedStrings)
{
	const char* fname	= "/tmp/CsvQuotedStringsTest.csv";
	const char* names[]	= { "line", "verse" };
	const char* types[]	= { "int32", "str" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << names[0] << ',' << names[1] << ',' << '\n';
	outfile << "10,`abc,\ndef, ghi`" << '\n';
	outfile << "20,`jkl, ``mno``, pqr`" << '\n';
	outfile << "30,stu `vwx` yz" << '\n';
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols		= std::extent<decltype(names)>::value;
		args.names			= names;
		args.dtype			= types;
		args.delimiter		= ',';
		args.lineterminator = '\n';
		args.quotechar		= '`';
		args.quoting		= true;	// strip outermost quotechar
		args.doublequote	= true;	// replace double quotechar with single
		args.skiprows		= 1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		// No filtering of any columns
		EXPECT_EQ( args.num_cols_out, args.num_cols );

		// Check the parsed string column metadata
		ASSERT_EQ( args.data[1]->dtype, GDF_STRING );
		auto stringList = reinterpret_cast<NVStrings*>(args.data[1]->data);

		ASSERT_NE( stringList, nullptr );
		auto stringCount = stringList->size();
		ASSERT_EQ( stringCount, 3u );
		auto stringLengths = std::unique_ptr<int[]>{ new int[stringCount] };
		ASSERT_NE( stringList->len(stringLengths.get(), false), 0u );

		// Check the actual strings themselves
		auto strings = std::unique_ptr<char*[]>{ new char*[stringCount] };
		for (size_t i = 0; i < stringCount; ++i) {
			ASSERT_GT( stringLengths[i], 0 );
			strings[i] = new char[stringLengths[i]];
		}
		EXPECT_EQ( stringList->to_host(strings.get(), 0, stringCount), 0 );
		EXPECT_STREQ( strings[0], "abc,\ndef, ghi" );
		EXPECT_STREQ( strings[1], "jkl, `mno`, pqr" );
		EXPECT_STREQ( strings[2], "stu `vwx` yz" );
		for (size_t i = 0; i < stringCount; ++i) {
			delete[] strings[i];
		}
	}
}

TEST(gdf_csv_test, KeepFullQuotedStrings)
{
	const char* fname	= "/tmp/CsvKeepFullQuotedStringsTest.csv";
	const char* names[]	= { "line", "verse" };
	const char* types[]	= { "int32", "str" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << names[0] << ',' << names[1] << ',' << '\n';
	outfile << "10,\"abc,\ndef, ghi\"" << '\n';
	outfile << "20,\"jkl, \"\"mno\"\", pqr\"" << '\n';
	outfile << "30,stu \"vwx\" yz" << '\n';
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols		= std::extent<decltype(names)>::value;
		args.names			= names;
		args.dtype			= types;
		args.delimiter		= ',';
		args.lineterminator = '\n';
		args.quotechar		= '\"';
		args.quoting		= false;	// do not strip outermost quotechar
		args.doublequote	= false;	// do not replace double quotechar with single
		args.skiprows		= 1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		// No filtering of any columns
		EXPECT_EQ( args.num_cols_out, args.num_cols );

		// Check the parsed string column metadata
		ASSERT_EQ( args.data[1]->dtype, GDF_STRING );
		auto stringList = reinterpret_cast<NVStrings*>(args.data[1]->data);

		ASSERT_NE( stringList, nullptr );
		auto stringCount = stringList->size();
		ASSERT_EQ( stringCount, 3u );
		auto stringLengths = std::unique_ptr<int[]>{ new int[stringCount] };
		ASSERT_NE( stringList->len(stringLengths.get(), false), 0u );

		// Check the actual strings themselves
		auto strings = std::unique_ptr<char*[]>{ new char*[stringCount] };
		for (size_t i = 0; i < stringCount; ++i) {
			ASSERT_GT( stringLengths[i], 0 );
			strings[i] = new char[stringLengths[i]];
		}
		EXPECT_EQ( stringList->to_host(strings.get(), 0, stringCount), 0 );
		EXPECT_STREQ( strings[0], "\"abc,\ndef, ghi\"" );
		EXPECT_STREQ( strings[1], "\"jkl, \"\"mno\"\", pqr\"" );
		EXPECT_STREQ( strings[2], "stu \"vwx\" yz" );
		for (size_t i = 0; i < stringCount; ++i) {
			delete[] strings[i];
		}
	}
}

TEST(gdf_csv_test, SpecifiedBoolValues)
{
	const char* fname			= "/tmp/CsvSpecifiedBoolValuesTest.csv";
	const char* names[]			= { "A", "B", "C" };
	const char* types[]			= { "int32", "int32", "short" };
	const char* trueValues[]	= { "yes", "Yes", "YES", "foo", "FOO" };
	const char* falseValues[]	= { "no", "No", "NO", "Bar", "bar" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << "YES,1,bar\nno,2,FOO\nBar,3,yes\nNo,4,NO\nYes,5,foo\n";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form		= gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer		= fname;
		args.num_cols			= std::extent<decltype(names)>::value;
		args.names				= names;
		args.dtype				= types;
		args.delimiter			= ',';
		args.lineterminator 	= '\n';
		args.true_values		= trueValues;
		args.num_true_values	= std::extent<decltype(trueValues)>::value;
		args.false_values		= falseValues;
		args.num_false_values	= std::extent<decltype(falseValues)>::value;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		// Booleans are the same (integer) data type, but valued at 0 or 1
		EXPECT_EQ( args.num_cols_out, args.num_cols );
		ASSERT_EQ( args.data[0]->dtype, GDF_INT32 );
		ASSERT_EQ( args.data[2]->dtype, GDF_INT16 );

		auto firstCol = gdf_host_column<int32_t>(args.data[0]);
		EXPECT_THAT(firstCol.hostdata(), ::testing::ElementsAre(1, 0, 0, 0, 1));
		auto thirdCol = gdf_host_column<int16_t>(args.data[2]);
		EXPECT_THAT(thirdCol.hostdata(), ::testing::ElementsAre(0, 1, 1, 0, 1));
	}
}

TEST(gdf_csv_test, Dates)
{
	const char* fname			= "/tmp/CsvDatesTest.csv";
	const char* names[]			= { "A" };
	const char* types[]			= { "date" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << "05/03/2001\n31/10/2010\n20/10/1994\n18/10/1990\n1/1/1970\n";
	outfile << "18/04/1995\n14/07/1994\n07/06/2006\n16/09/2005\n2/2/1970\n";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form	= gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer	= fname;
		args.num_cols			= std::extent<decltype(names)>::value;
		args.names				= names;
		args.dtype				= types;
		args.delimiter			= ',';
		args.lineterminator 	= '\n';
		args.dayfirst			= true;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		EXPECT_EQ( args.num_cols_out, args.num_cols );
		ASSERT_EQ( args.data[0]->dtype, GDF_DATE64 );

		auto ACol = gdf_host_column<uint64_t>(args.data[0]);
		EXPECT_THAT( ACol.hostdata(),
			::testing::ElementsAre(983750400000, 1288483200000, 782611200000,
								   656208000000, 0, 798163200000, 774144000000,
								   1149638400000, 1126828800000, 2764800000) );
	}
}



TEST(gdf_csv_float_test, FloatTest)
{
	const char* fname			= "/tmp/CsvFloatTest.csv";
	const char* names[]			= { "A" };
	const char* types[]			= { "float64" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << "10.5\n12.0\n123.123\n1234.1234\n";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form	= gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer	= fname;
		args.num_cols			= std::extent<decltype(names)>::value;
		args.names				= names;
		args.dtype				= types;
		args.delimiter			= ',';
		args.lineterminator 	= '\n';
		args.decimal='.';
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		EXPECT_EQ( args.num_cols_out, args.num_cols );
		ASSERT_EQ( args.data[0]->dtype, GDF_FLOAT64 );


		auto ACol = gdf_host_column<double>(args.data[0]);
		std::vector<double> test_values({10.5,12.0,123.123,1234.1234});
		compare_floats(ACol,test_values);

	}
}


TEST(gdf_csv_type_inference_test, InferenceTest)
{
	const char* fname			= "/tmp/CsvInferenceTest.csv";
	const char* names[]			= { "A", "B" };
	const char* types[]			= { "float64", "int32" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << "1.1,1\n12.12,2\n123.123,3\n1234.1234,4\n";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form	= gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer	= fname;
		args.num_cols			= std::extent<decltype(names)>::value;
		args.names				= names;
		args.dtype				= types;
		args.delimiter			= ',';
		args.lineterminator 	= '\n';
		args.decimal='.';
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		EXPECT_EQ( args.num_cols_out, args.num_cols );
		ASSERT_EQ( args.data[0]->dtype, GDF_FLOAT64 );
		ASSERT_EQ( args.data[1]->dtype, GDF_INT32 );

		auto ACol = gdf_host_column<double>(args.data[0]);
		std::vector<double> test_values({1.1,12.12,123.123,1234.1234});
		compare_floats(ACol,test_values);

		auto BCol = gdf_host_column<int>(args.data[1]);
		EXPECT_THAT( BCol.hostdata(),
				::testing::ElementsAre(1,2,3,4) );

		csv_read_arg args_inferred{};
		args_inferred.input_data_form	= gdf_csv_input_form::FILE_PATH;
		args_inferred.filepath_or_buffer	= fname;
		args_inferred.num_cols			= std::extent<decltype(names)>::value;
		args_inferred.names				= names;
		args_inferred.dtype				= nullptr;
		args_inferred.delimiter			= ',';
		args_inferred.lineterminator 	= '\n';
		args_inferred.decimal='.';
		EXPECT_EQ( read_csv(&args_inferred), GDF_SUCCESS );


		EXPECT_EQ( args_inferred.num_cols_out, args_inferred.num_cols );
		ASSERT_EQ( args_inferred.data[0]->dtype, GDF_FLOAT64 );
		ASSERT_EQ( args_inferred.data[1]->dtype, GDF_INT64 );

		auto AColInferred = gdf_host_column<double>(args_inferred.data[0]);
		compare_floats(AColInferred,test_values);

		auto BColInferred = gdf_host_column<int64_t>(args_inferred.data[1]);
		EXPECT_THAT( BColInferred.hostdata(),
				::testing::ElementsAre(1,2,3,4) );




	}
}

TEST(gdf_csv_type_inference_header_types_test, HeaderInferenceTest)
{
	const char* fname			= "/tmp/CsvInferenceTest.csv";
	const char* names[]			= { "A", "B" };
	const char* types[]			= { "float64", "int32" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << "float64,int32\n1.1,1\n12.12,2\n123.123,3\n1234.1234,4\n";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form	= gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer	= fname;
		args.num_cols			= std::extent<decltype(names)>::value;

		args.dtype				= types;

		args.delimiter			= ',';
		args.lineterminator 	= '\n';
		args.decimal='.';
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		EXPECT_EQ( args.num_cols_out, args.num_cols );
		ASSERT_EQ( args.data[0]->dtype, GDF_FLOAT64 );
		ASSERT_EQ( args.data[1]->dtype, GDF_INT32 );

		auto ACol = gdf_host_column<double>(args.data[0]);
		std::vector<double> test_values({1.1,12.12,123.123,1234.1234});
		compare_floats(ACol,test_values);

		auto BCol = gdf_host_column<int>(args.data[1]);
		EXPECT_THAT( BCol.hostdata(),
				::testing::ElementsAre(1,2,3,4) );

		csv_read_arg args_inferred{};
		args_inferred.input_data_form	= gdf_csv_input_form::FILE_PATH;
		args_inferred.filepath_or_buffer	= fname;
		args_inferred.num_cols			= std::extent<decltype(names)>::value;
		args_inferred.header = 0;
		args_inferred.delimiter			= ',';
		args_inferred.lineterminator 	= '\n';
		args_inferred.decimal='.';
		EXPECT_EQ( read_csv(&args_inferred), GDF_SUCCESS );


		EXPECT_EQ( args_inferred.num_cols_out, args_inferred.num_cols );
		ASSERT_EQ( args_inferred.data[0]->dtype, GDF_FLOAT64 );
		ASSERT_EQ( args_inferred.data[1]->dtype, GDF_INT64 );

		auto AColInferred = gdf_host_column<double>(args_inferred.data[0]);
		compare_floats(AColInferred,test_values);

		auto BColInferred = gdf_host_column<int64_t>(args_inferred.data[1]);
		EXPECT_THAT( BColInferred.hostdata(),
				::testing::ElementsAre(1,2,3,4) );




	}
}


TEST(gdf_csv_type_inference_header_from_buffer_test, BufferHeaderInferenceTest)
{
	const char* fname			= "float64,int32\n1.1,1\n12.12,2\n123.123,3\n1234.1234,4\n";
	const char* names[]			= { "A", "B" };
	const char* types[]			= { "float64", "int32" };
		csv_read_arg args{};
		args.input_data_form	= gdf_csv_input_form::HOST_BUFFER;
		args.filepath_or_buffer	= fname;
		args.buffer_size = sizeof("float64,int32\n1.1,1\n12.12,2\n123.123,3\n1234.1234,4\n");
		args.num_cols			= std::extent<decltype(names)>::value;
		args.dtype				= types;
		args.delimiter			= ',';
		args.lineterminator 	= '\n';
		args.decimal='.';
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		EXPECT_EQ( args.num_cols_out, args.num_cols );
		ASSERT_EQ( args.data[0]->dtype, GDF_FLOAT64 );
		ASSERT_EQ( args.data[1]->dtype, GDF_INT32 );

		auto ACol = gdf_host_column<double>(args.data[0]);
		std::vector<double> test_values({1.1,12.12,123.123,1234.1234});
		compare_floats(ACol,test_values);

		auto BCol = gdf_host_column<int>(args.data[1]);
		EXPECT_THAT( BCol.hostdata(),
				::testing::ElementsAre(1,2,3,4) );

		csv_read_arg args_inferred{};
		args_inferred.input_data_form	= gdf_csv_input_form::HOST_BUFFER;
		args_inferred.filepath_or_buffer	= fname;
		args_inferred.num_cols			= std::extent<decltype(names)>::value;
		args_inferred.buffer_size = sizeof("float64,int32\n1.1,1\n12.12,2\n123.123,3\n1234.1234,4\n");
		args_inferred.delimiter			= ',';
		args_inferred.lineterminator 	= '\n';
		args_inferred.decimal='.';
		EXPECT_EQ( read_csv(&args_inferred), GDF_SUCCESS );


		EXPECT_EQ( args_inferred.num_cols_out, args_inferred.num_cols );
		ASSERT_EQ( args_inferred.data[0]->dtype, GDF_FLOAT64 );
		ASSERT_EQ( args_inferred.data[1]->dtype, GDF_INT64 );

		auto AColInferred = gdf_host_column<double>(args_inferred.data[0]);
		compare_floats(AColInferred,test_values);

		auto BColInferred = gdf_host_column<int64_t>(args_inferred.data[1]);
		EXPECT_THAT( BColInferred.hostdata(),
				::testing::ElementsAre(1,2,3,4) );




}

